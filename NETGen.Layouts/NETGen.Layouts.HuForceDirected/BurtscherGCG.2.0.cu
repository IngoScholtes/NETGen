#include "hip/hip_runtime.h"
/*
CUDA BarnesHut v2.0: Simulation of the gravitational forces
in a galactic cluster using the Barnes-Hut n-body algorithm

Copyright (c) 2011, Texas State University-San Marcos.  All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

   * Redistributions of source code must retain the above copyright notice, 
     this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.
   * Neither the name of Texas State University-San Marcos nor the names of its
     contributors may be used to endorse or promote products derived from this
     software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/


#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


// thread count
#define THREADS1 512  /* must be a power of 2 */
#define THREADS2 1024
#define THREADS3 1024
#define THREADS4 256
#define THREADS5 256
#define THREADS6 512

// block count = factor * #SMs
#define FACTOR1 3
#define FACTOR2 1
#define FACTOR3 1  /* must all be resident at the same time */
#define FACTOR4 1  /* must all be resident at the same time */
#define FACTOR5 5
#define FACTOR6 3

#define WARPSIZE 32
#define MAXDEPTH 32


/******************************************************************************/

// childd is aliased with velxd, velyd, velzd, accxd, accyd, acczd, and sortd but they never use the same memory locations
__constant__ int nnodesd, nbodiesd;
__constant__ float dtimed, dthfd, epssqd, itolsqd;
__constant__ volatile float *massd, *posxd, *posyd, *poszd, *velxd, *velyd, *velzd, *accxd, *accyd, *acczd;
__constant__ volatile float *maxxd, *maxyd, *maxzd, *minxd, *minyd, *minzd;
__constant__ volatile int *errd, *sortd, *childd, *countd, *startd;

__device__ volatile int stepd, bottomd, maxdepthd, blkcntd;
__device__ volatile float radiusd;


/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void InitializationKernel()
{
  *errd = 0;
  stepd = -1;
  maxdepthd = 1;
  blkcntd = 0;
}


/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS1, FACTOR1)
void BoundingBoxKernel()
{
  register int i, j, k, inc;
  register float val, minx, maxx, miny, maxy, minz, maxz;
  __shared__ volatile float sminx[THREADS1], smaxx[THREADS1], sminy[THREADS1], smaxy[THREADS1], sminz[THREADS1], smaxz[THREADS1];

  // initialize with valid data (in case #bodies < #threads)
  minx = maxx = posxd[0];
  miny = maxy = posyd[0];
  minz = maxz = poszd[0];

  // scan all bodies
  i = threadIdx.x;
  inc = THREADS1 * gridDim.x;
  for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc) {
    val = posxd[j];
    minx = min(minx, val);
    maxx = max(maxx, val);
    val = posyd[j];
    miny = min(miny, val);
    maxy = max(maxy, val);
    val = poszd[j];
    minz = min(minz, val);
    maxz = max(maxz, val);
  }

  // reduction in shared memory
  sminx[i] = minx;
  smaxx[i] = maxx;
  sminy[i] = miny;
  smaxy[i] = maxy;
  sminz[i] = minz;
  smaxz[i] = maxz;

  for (j = THREADS1 / 2; j > 0; j /= 2) {
    __syncthreads();
    if (i < j) {
      k = i + j;
      sminx[i] = minx = min(minx, sminx[k]);
      smaxx[i] = maxx = max(maxx, smaxx[k]);
      sminy[i] = miny = min(miny, sminy[k]);
      smaxy[i] = maxy = max(maxy, smaxy[k]);
      sminz[i] = minz = min(minz, sminz[k]);
      smaxz[i] = maxz = max(maxz, smaxz[k]);
    }
  }

  // write block result to global memory
  if (i == 0) {
    k = blockIdx.x;
    minxd[k] = minx;
    maxxd[k] = maxx;
    minyd[k] = miny;
    maxyd[k] = maxy;
    minzd[k] = minz;
    maxzd[k] = maxz;
    __threadfence();

    inc = gridDim.x - 1;
    if (inc == atomicInc((unsigned int *)&blkcntd, inc)) {
      // I'm the last block, so combine all block results
      for (j = 0; j <= inc; j++) {
        minx = min(minx, minxd[j]);
        maxx = max(maxx, maxxd[j]);
        miny = min(miny, minyd[j]);
        maxy = max(maxy, maxyd[j]);
        minz = min(minz, minzd[j]);
        maxz = max(maxz, maxzd[j]);
      }

      // compute 'radius'
      val = max(maxx - minx, maxy - miny);
      radiusd = max(val, maxz - minz) * 0.5f;

      // create root node
      k = nnodesd;
      bottomd = k;

      massd[k] = -1.0f;
      startd[k] = 0;
      posxd[k] = (minx + maxx) * 0.5f;
      posyd[k] = (miny + maxy) * 0.5f;
      poszd[k] = (minz + maxz) * 0.5f;
      k *= 8;
      for (i = 0; i < 8; i++) childd[k + i] = -1;

      stepd++;
    }
  }
}


/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS2, FACTOR2)
void TreeBuildingKernel()
{
  register int i, j, k, depth, localmaxdepth, skip, inc;
  register float x, y, z, r;
  register float px, py, pz;
  register int ch, n, cell, locked, patch;
  register float radius, rootx, rooty, rootz;

  // cache root data
  radius = radiusd;
  rootx = posxd[nnodesd];
  rooty = posyd[nnodesd];
  rootz = poszd[nnodesd];

  localmaxdepth = 1;
  skip = 1;
  inc = blockDim.x * gridDim.x;
  i = threadIdx.x + blockIdx.x * blockDim.x;

  // iterate over all bodies assigned to thread
  while (i < nbodiesd) {
    if (skip != 0) {
      // new body, so start traversing at root
      skip = 0;
      px = posxd[i];
      py = posyd[i];
      pz = poszd[i];
      n = nnodesd;
      depth = 1;
      r = radius;
      j = 0;
      // determine which child to follow
      if (rootx < px) j = 1;
      if (rooty < py) j += 2;
      if (rootz < pz) j += 4;
    }

    // follow path to leaf cell
    ch = childd[n*8+j];
    while (ch >= nbodiesd) {
      n = ch;
      depth++;
      r *= 0.5f;
      j = 0;
      // determine which child to follow
      if (posxd[n] < px) j = 1;
      if (posyd[n] < py) j += 2;
      if (poszd[n] < pz) j += 4;
      ch = childd[n*8+j];
    }

    if (ch != -2) {  // skip if child pointer is locked and try again later
      locked = n*8+j;
      if (ch == atomicCAS((int *)&childd[locked], ch, -2)) {  // try to lock
        if (ch == -1) {
          // if null, just insert the new body
          childd[locked] = i;
        } else {  // there already is a body in this position
          patch = -1;
          // create new cell(s) and insert the old and new body
          do {
            depth++;

            cell = atomicSub((int *)&bottomd, 1) - 1;
            if (cell <= nbodiesd) {
              *errd = 1;
              bottomd = nnodesd;
            }
            patch = max(patch, cell);

            x = (j & 1) * r;
            y = ((j >> 1) & 1) * r;
            z = ((j >> 2) & 1) * r;
            r *= 0.5f;

            massd[cell] = -1.0f;
            startd[cell] = -1;
            x = posxd[cell] = posxd[n] - r + x;
            y = posyd[cell] = posyd[n] - r + y;
            z = poszd[cell] = poszd[n] - r + z;
            for (k = 0; k < 8; k++) childd[cell*8+k] = -1;

            if (patch != cell) { 
              childd[n*8+j] = cell;
            }

            j = 0;
            if (x < posxd[ch]) j = 1;
            if (y < posyd[ch]) j += 2;
            if (z < poszd[ch]) j += 4;
            childd[cell*8+j] = ch;

            n = cell;
            j = 0;
            if (x < px) j = 1;
            if (y < py) j += 2;
            if (z < pz) j += 4;

            ch = childd[n*8+j];
            // repeat until the two bodies are different children
          } while (ch >= 0);
          childd[n*8+j] = i;
          __threadfence();  // push out subtree
          childd[locked] = patch;
        }
        __threadfence();  // push out results

        localmaxdepth = max(depth, localmaxdepth);
        i += inc;  // move on to next body
        skip = 1;
      }
    }
    __syncthreads();  // throttle
  }
  // record maximum tree depth
  atomicMax((int *)&maxdepthd, localmaxdepth);
}


/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS3, FACTOR3)
void SummarizationKernel()
{
  register int i, j, k, ch, inc, missing, cnt, bottom;
  register float m, cm, px, py, pz;
  __shared__ volatile int child[THREADS3 * 8];

  bottom = bottomd;
  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  missing = 0;
  // iterate over all cells assigned to thread
  while (k <= nnodesd) {
    if (missing == 0) {
      // new cell, so initialize
      cm = 0.0f;
      px = 0.0f;
      py = 0.0f;
      pz = 0.0f;
      cnt = 0;
      j = 0;
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= 0) {
          if (i != j) {
            // move children to front (needed later for speed)
            childd[k*8+i] = -1;
            childd[k*8+j] = ch;
          }
          child[missing*THREADS3+threadIdx.x] = ch;  // cache missing children
          m = massd[ch];
          missing++;
          if (m >= 0.0f) {
            // child is ready
            missing--;
            if (ch >= nbodiesd) {  // count bodies (needed later)
              cnt += countd[ch] - 1;
            }
            // add child's contribution
            cm += m;
            px += posxd[ch] * m;
            py += posyd[ch] * m;
            pz += poszd[ch] * m;
          }
          j++;
        }
      }
      __threadfence();  // for performance only
      cnt += j;
    }

    if (missing != 0) {
      do {
        // poll missing child
        ch = child[(missing-1)*THREADS3+threadIdx.x];
        m = massd[ch];
        if (m >= 0.0f) {
          // child is now ready
          missing--;
          if (ch >= nbodiesd) {
            // count bodies (needed later)
            cnt += countd[ch] - 1;
          }
          // add child's contribution
          cm += m;
          px += posxd[ch] * m;
          py += posyd[ch] * m;
          pz += poszd[ch] * m;
        }
        // repeat until we are done or child is not ready
      } while ((m >= 0.0f) && (missing != 0));
    }

    if (missing == 0) {
      // all children are ready, so store computed information
      countd[k] = cnt;
      m = 1.0f / cm;
      posxd[k] = px * m;
      posyd[k] = py * m;
      poszd[k] = pz * m;
      __threadfence();  // make sure data are visible before setting mass
      massd[k] = cm;
      __threadfence();  // push out results
      k += inc;  // move on to next cell
    }
  }
}


/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS4, FACTOR4)
void SortKernel()
{
  register int i, k, ch, dec, start, bottom;

  bottom = bottomd;
  dec = blockDim.x * gridDim.x;
  k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

  // iterate over all cells assigned to thread
  while (k >= bottom) {
    start = startd[k];
    if (start >= 0) {
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= nbodiesd) {
          // child is a cell
          startd[ch] = start;  // set start ID of child
          start += countd[ch];  // add #bodies in subtree
        } else if (ch >= 0) {
          // child is a body
          sortd[start] = ch;  // record body in 'sorted' array
          start++;
        }
      }
      k -= dec;  // move on to next cell
    }
    __syncthreads();  // throttle
  }
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS5, FACTOR5)
void ForceCalculationKernel()
{
  register int i, j, k, n, depth, base, sbase, diff;
  register float px, py, pz, ax, ay, az, dx, dy, dz, tmp;
  __shared__ volatile int pos[MAXDEPTH * THREADS5/WARPSIZE], node[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ volatile float dq[MAXDEPTH * THREADS5/WARPSIZE];
  __shared__ volatile int step, maxdepth;

  if (0 == threadIdx.x) {
    step = stepd;
    maxdepth = maxdepthd;
    tmp = radiusd;
    // precompute values that depend only on tree level
    dq[0] = tmp * tmp * itolsqd;
    for (i = 1; i < maxdepth; i++) {
      dq[i] = dq[i - 1] * 0.25f;
    }

    if (maxdepth > MAXDEPTH) {
      *errd = maxdepth;
    }
  }
  __syncthreads();

  if (maxdepth <= MAXDEPTH) {
    // figure out first thread in each warp (lane 0)
    base = threadIdx.x / WARPSIZE;
    sbase = base * WARPSIZE;
    j = base * MAXDEPTH;

    diff = threadIdx.x - sbase;
    // make multiple copies to avoid index calculations later
    if (diff < MAXDEPTH) {
      dq[diff+j] = dq[diff];
    }
    __syncthreads();

    // iterate over all bodies assigned to thread
    for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
      i = sortd[k];  // get permuted/sorted index
      // cache position info
      px = posxd[i];
      py = posyd[i];
      pz = poszd[i];

      ax = 0.0f;
      ay = 0.0f;
      az = 0.0f;

      // initialize iteration stack, i.e., push root node onto stack
      depth = j;
      if (sbase == threadIdx.x) {
        node[j] = nnodesd;
        pos[j] = 0;
      }
      __threadfence();  // make sure it's visible

      while (depth >= j) {
        // stack is not empty
        while (pos[depth] < 8) {
          // node on top of stack has more children to process
          n = childd[node[depth]*8+pos[depth]];  // load child pointer
          if (sbase == threadIdx.x) {
            // I'm the first thread in the warp
            pos[depth]++;
          }
          __threadfence();  // make sure it's visible
          if (n >= 0) {
            dx = posxd[n] - px;
            dy = posyd[n] - py;
            dz = poszd[n] - pz;
            tmp = dx*dx + (dy*dy + (dz*dz + epssqd));  // compute distance squared (plus softening)
            if ((n < nbodiesd) || __all(tmp >= dq[depth])) {  // check if all threads agree that cell is far enough away (or is a body)
              tmp = rsqrtf(tmp);  // compute distance
              tmp = massd[n] * tmp * tmp * tmp;
              ax += dx * tmp;
              ay += dy * tmp;
              az += dz * tmp;
            } else {
              // push cell onto stack
              depth++;
              if (sbase == threadIdx.x) {
                node[depth] = n;
                pos[depth] = 0;
              }
              __threadfence();  // make sure it's visible
            }
          } else {
            depth = max(j, depth - 1);  // early out because all remaining children are also zero
          }
        }
        depth--;  // done with this level
      }

      if (step > 0) {
        // update velocity
        velxd[i] += (ax - accxd[i]) * dthfd;
        velyd[i] += (ay - accyd[i]) * dthfd;
        velzd[i] += (az - acczd[i]) * dthfd;
      }

      // save computed acceleration
      accxd[i] = ax;
      accyd[i] = ay;
      acczd[i] = az;
    }
  }
}


/******************************************************************************/
/*** advance bodies ***********************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS6, FACTOR6)
void IntegrationKernel()
{
  register int i, inc;
  register float dvelx, dvely, dvelz;
  register float velhx, velhy, velhz;

  // iterate over all bodies assigned to thread
  inc = blockDim.x * gridDim.x;
  for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
    // integrate
    dvelx = accxd[i] * dthfd;
    dvely = accyd[i] * dthfd;
    dvelz = acczd[i] * dthfd;

    velhx = velxd[i] + dvelx;
    velhy = velyd[i] + dvely;
    velhz = velzd[i] + dvelz;

    posxd[i] += velhx * dtimed;
    posyd[i] += velhy * dtimed;
    poszd[i] += velhz * dtimed;

    velxd[i] = velhx + dvelx;
    velyd[i] = velhy + dvely;
    velzd[i] = velhz + dvelz;
  }
}


/******************************************************************************/

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}


/******************************************************************************/

// random number generator

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

static int A = 1;
static int B = 0;
static int randx = 1;
static int lastrand;


static void drndset(int seed)
{
   A = 1;
   B = 0;
   randx = (A * seed + B) & MASK;
   A = (MULT * A) & MASK;
   B = (MULT * B + ADD) & MASK;
}


static double drnd()
{
   lastrand = randx;
   randx = (A * randx + B) & MASK;
   return (double)lastrand / TWOTO31;
}


/******************************************************************************/

int main(int argc, char *argv[])
{
  register int i, run, blocks;
  register int nnodes, nbodies, step, timesteps;
  register int runtime, mintime;
  int error;
  register float dtime, dthf, epssq, itolsq;
  float time, timing[7];
  clock_t starttime, endtime;
  hipEvent_t start, stop;
  float *mass, *posx, *posy, *posz, *velx, *vely, *velz;

  int *errl, *sortl, *childl, *countl, *startl;
  float *massl;
  float *posxl, *posyl, *poszl;
  float *velxl, *velyl, *velzl;
  float *accxl, *accyl, *acczl;
  float *maxxl, *maxyl, *maxzl;
  float *minxl, *minyl, *minzl;
  register double rsc, vsc, r, v, x, y, z, sq, scale;

  // perform some checks

  fprintf(stderr, "CUDA BarnesHut v2.0\n");
  if (argc != 3) {
    fprintf(stderr, "\n");
    fprintf(stderr, "arguments: number_of_bodies number_of_timesteps\n");
    exit(-1);
  }

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "There is no device supporting CUDA\n");
    exit(-1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "There is no CUDA capable device\n");
    exit(-1);
  }
  if (deviceProp.major < 2) {
    fprintf(stderr, "Need at least compute capability 2.0\n");
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }

  blocks = deviceProp.multiProcessorCount;
  fprintf(stderr, "blocks = %d\n", blocks);

  if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE-1) != 0)) {
    fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
    exit(-1);
  }
  if (MAXDEPTH > WARPSIZE) {
    fprintf(stderr, "MAXDEPTH must be less than or equal to WARPSIZE\n");
    exit(-1);
  }
  if ((THREADS1 <= 0) || (THREADS1 & (THREADS1-1) != 0)) {
    fprintf(stderr, "THREADS1 must be greater than zero and a power of two\n");
    exit(-1);
  }

  // set L1/shared memory configuration
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(IntegrationKernel), hipFuncCachePreferL1);

  hipGetLastError();  // reset error value
  for (run = 0; run < 3; run++) {
    for (i = 0; i < 7; i++) timing[i] = 0.0f;

    nbodies = atoi(argv[1]);
    if (nbodies < 1) {
      fprintf(stderr, "nbodies is too small: %d\n", nbodies);
      exit(-1);
    }
    if (nbodies > (1 << 30)) {
      fprintf(stderr, "nbodies is too large: %d\n", nbodies);
      exit(-1);
    }
    nnodes = nbodies * 2;
    if (nnodes < 1024*blocks) nnodes = 1024*blocks;
    while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
    nnodes--;

    timesteps = atoi(argv[2]);
    dtime = 0.025;  dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;
    itolsq = 1.0f / (0.5 * 0.5);

    // allocate memory

    if (run == 0) {
      fprintf(stderr, "nodes = %d\n", nnodes+1);
      fprintf(stderr, "configuration: %d bodies, %d time steps\n", nbodies, timesteps);

      mass = (float *)malloc(sizeof(float) * nbodies);
      if (mass == NULL) {fprintf(stderr, "cannot allocate mass\n");  exit(-1);}
      posx = (float *)malloc(sizeof(float) * nbodies);
      if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
      posy = (float *)malloc(sizeof(float) * nbodies);
      if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
      posz = (float *)malloc(sizeof(float) * nbodies);
      if (posz == NULL) {fprintf(stderr, "cannot allocate posz\n");  exit(-1);}
      velx = (float *)malloc(sizeof(float) * nbodies);
      if (velx == NULL) {fprintf(stderr, "cannot allocate velx\n");  exit(-1);}
      vely = (float *)malloc(sizeof(float) * nbodies);
      if (vely == NULL) {fprintf(stderr, "cannot allocate vely\n");  exit(-1);}
      velz = (float *)malloc(sizeof(float) * nbodies);
      if (velz == NULL) {fprintf(stderr, "cannot allocate velz\n");  exit(-1);}

      if (hipSuccess != hipMalloc((void **)&errl, sizeof(int))) fprintf(stderr, "could not allocate errd\n");  CudaTest("couldn't allocate errd");
      if (hipSuccess != hipMalloc((void **)&childl, sizeof(int) * (nnodes+1) * 8)) fprintf(stderr, "could not allocate childd\n");  CudaTest("couldn't allocate childd");
      if (hipSuccess != hipMalloc((void **)&massl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate massd\n");  CudaTest("couldn't allocate massd");
      if (hipSuccess != hipMalloc((void **)&posxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posxd\n");  CudaTest("couldn't allocate posxd");
      if (hipSuccess != hipMalloc((void **)&posyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posyd\n");  CudaTest("couldn't allocate posyd");
      if (hipSuccess != hipMalloc((void **)&poszl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate poszd\n");  CudaTest("couldn't allocate poszd");
      if (hipSuccess != hipMalloc((void **)&countl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate countd\n");  CudaTest("couldn't allocate countd");
      if (hipSuccess != hipMalloc((void **)&startl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate startd\n");  CudaTest("couldn't allocate startd");

      // alias arrays
      int inc = (nbodies + WARPSIZE - 1) & (-WARPSIZE);
      velxl = (float *)&childl[0*inc];
      velyl = (float *)&childl[1*inc];
      velzl = (float *)&childl[2*inc];
      accxl = (float *)&childl[3*inc];
      accyl = (float *)&childl[4*inc];
      acczl = (float *)&childl[5*inc];
      sortl = (int *)&childl[6*inc];

      if (hipSuccess != hipMalloc((void **)&maxxl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate maxxd\n");  CudaTest("couldn't allocate maxxd");
      if (hipSuccess != hipMalloc((void **)&maxyl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate maxyd\n");  CudaTest("couldn't allocate maxyd");
      if (hipSuccess != hipMalloc((void **)&maxzl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate maxzd\n");  CudaTest("couldn't allocate maxzd");
      if (hipSuccess != hipMalloc((void **)&minxl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate minxd\n");  CudaTest("couldn't allocate minxd");
      if (hipSuccess != hipMalloc((void **)&minyl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate minyd\n");  CudaTest("couldn't allocate minyd");
      if (hipSuccess != hipMalloc((void **)&minzl, sizeof(float) * blocks)) fprintf(stderr, "could not allocate minzd\n");  CudaTest("couldn't allocate minzd");

      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(nnodesd), &nnodes, sizeof(int))) fprintf(stderr, "copying of nnodes to device failed\n");  CudaTest("nnode copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int))) fprintf(stderr, "copying of nbodies to device failed\n");  CudaTest("nbody copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(errd), &errl, sizeof(int))) fprintf(stderr, "copying of err to device failed\n");  CudaTest("err copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dtimed), &dtime, sizeof(float))) fprintf(stderr, "copying of dtime to device failed\n");  CudaTest("dtime copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dthfd), &dthf, sizeof(float))) fprintf(stderr, "copying of dthf to device failed\n");  CudaTest("dthf copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float))) fprintf(stderr, "copying of epssq to device failed\n");  CudaTest("epssq copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float))) fprintf(stderr, "copying of itolsq to device failed\n");  CudaTest("itolsq copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(sortd), &sortl, sizeof(int))) fprintf(stderr, "copying of sortl to device failed\n");  CudaTest("sortl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(countd), &countl, sizeof(int))) fprintf(stderr, "copying of countl to device failed\n");  CudaTest("countl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(startd), &startl, sizeof(int))) fprintf(stderr, "copying of startl to device failed\n");  CudaTest("startl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(childd), &childl, sizeof(int))) fprintf(stderr, "copying of childl to device failed\n");  CudaTest("childl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(massd), &massl, sizeof(int))) fprintf(stderr, "copying of massl to device failed\n");  CudaTest("massl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(posxd), &posxl, sizeof(int))) fprintf(stderr, "copying of posxl to device failed\n");  CudaTest("posxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(posyd), &posyl, sizeof(int))) fprintf(stderr, "copying of posyl to device failed\n");  CudaTest("posyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(poszd), &poszl, sizeof(int))) fprintf(stderr, "copying of poszl to device failed\n");  CudaTest("poszl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(velxd), &velxl, sizeof(int))) fprintf(stderr, "copying of velxl to device failed\n");  CudaTest("velxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(velyd), &velyl, sizeof(int))) fprintf(stderr, "copying of velyl to device failed\n");  CudaTest("velyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(velzd), &velzl, sizeof(int))) fprintf(stderr, "copying of velzl to device failed\n");  CudaTest("velzl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(accxd), &accxl, sizeof(int))) fprintf(stderr, "copying of accxl to device failed\n");  CudaTest("accxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(accyd), &accyl, sizeof(int))) fprintf(stderr, "copying of accyl to device failed\n");  CudaTest("accyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(acczd), &acczl, sizeof(int))) fprintf(stderr, "copying of acczl to device failed\n");  CudaTest("acczl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxxd), &maxxl, sizeof(int))) fprintf(stderr, "copying of maxxl to device failed\n");  CudaTest("maxxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxyd), &maxyl, sizeof(int))) fprintf(stderr, "copying of maxyl to device failed\n");  CudaTest("maxyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxzd), &maxzl, sizeof(int))) fprintf(stderr, "copying of maxzl to device failed\n");  CudaTest("maxzl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minxd), &minxl, sizeof(int))) fprintf(stderr, "copying of minxl to device failed\n");  CudaTest("minxl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minyd), &minyl, sizeof(int))) fprintf(stderr, "copying of minyl to device failed\n");  CudaTest("minyl copy to device failed");
      if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minzd), &minzl, sizeof(int))) fprintf(stderr, "copying of minzl to device failed\n");  CudaTest("minzl copy to device failed");
    }

    // generate input

    drndset(7);
    rsc = (3 * 3.1415926535897932384626433832795) / 16;
    vsc = sqrt(1.0 / rsc);
    for (i = 0; i < nbodies; i++) {
      mass[i] = 1.0 / nbodies;
      r = 1.0 / sqrt(pow(drnd()*0.999, -2.0/3.0) - 1);
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = rsc * r / sqrt(sq);
      posx[i] = x * scale;
      posy[i] = y * scale;
      posz[i] = z * scale;

      do {
        x = drnd();
        y = drnd() * 0.1;
      } while (y > x*x * pow(1 - x*x, 3.5));
      v = x * sqrt(2.0 / sqrt(1 + r*r));
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = vsc * v / sqrt(sq);
      velx[i] = x * scale;
      vely[i] = y * scale;
      velz[i] = z * scale;
    }

    if (hipSuccess != hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of mass to device failed\n");  CudaTest("mass copy to device failed");
    if (hipSuccess != hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posx to device failed\n");  CudaTest("posx copy to device failed");
    if (hipSuccess != hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posy to device failed\n");  CudaTest("posy copy to device failed");
    if (hipSuccess != hipMemcpy(poszl, posz, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posz to device failed\n");  CudaTest("posz copy to device failed");
    if (hipSuccess != hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velx to device failed\n");  CudaTest("velx copy to device failed");
    if (hipSuccess != hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of vely to device failed\n");  CudaTest("vely copy to device failed");
    if (hipSuccess != hipMemcpy(velzl, velz, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velz to device failed\n");  CudaTest("velz copy to device failed");

    // run timesteps (lauch GPU kernels)

    hipEventCreate(&start);  hipEventCreate(&stop);  
    starttime = clock();
    hipEventRecord(start, 0);
    InitializationKernel<<<1, 1>>>();
    hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    timing[0] += time;
    CudaTest("kernel 0 launch failed");

    for (step = 0; step < timesteps; step++) {
      hipEventRecord(start, 0);
      BoundingBoxKernel<<<blocks * FACTOR1, THREADS1>>>();
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[1] += time;
      CudaTest("kernel 1 launch failed");

      hipEventRecord(start, 0);
      TreeBuildingKernel<<<blocks * FACTOR2, THREADS2>>>();
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[2] += time;
      CudaTest("kernel 2 launch failed");

      hipEventRecord(start, 0);
      SummarizationKernel<<<blocks * FACTOR3, THREADS3>>>();
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[3] += time;
      CudaTest("kernel 3 launch failed");

      hipEventRecord(start, 0);
      SortKernel<<<blocks * FACTOR4, THREADS4>>>();
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[4] += time;
      CudaTest("kernel 4 launch failed");

      hipEventRecord(start, 0);
      ForceCalculationKernel<<<blocks * FACTOR5, THREADS5>>>();
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[5] += time;
      CudaTest("kernel 5 launch failed");

      hipEventRecord(start, 0);
      IntegrationKernel<<<blocks * FACTOR6, THREADS6>>>();
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[6] += time;
      CudaTest("kernel 6 launch failed");
    }
    endtime = clock();
    CudaTest("kernel launch failed");
    hipEventDestroy(start);  hipEventDestroy(stop);

    // transfer result back to CPU
    if (hipSuccess != hipMemcpy(&error, errl, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of err from device failed\n");  CudaTest("err copy from device failed");
    if (hipSuccess != hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posx from device failed\n");  CudaTest("posx copy from device failed");
    if (hipSuccess != hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posy from device failed\n");  CudaTest("posy copy from device failed");
    if (hipSuccess != hipMemcpy(posz, poszl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posz from device failed\n");  CudaTest("posz copy from device failed");
    if (hipSuccess != hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velx from device failed\n");  CudaTest("velx copy from device failed");
    if (hipSuccess != hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of vely from device failed\n");  CudaTest("vely copy from device failed");
    if (hipSuccess != hipMemcpy(velz, velzl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velz from device failed\n");  CudaTest("velz copy from device failed");

    runtime = (int) (1000.0f * (endtime - starttime) / CLOCKS_PER_SEC);
    fprintf(stderr, "runtime: %d ms  (", runtime);
    time = 0;
    for (i = 1; i < 7; i++) {
      fprintf(stderr, " %.1f ", timing[i]);
      time += timing[i];
    }
    if (error == 0) {
      fprintf(stderr, ") = %.1f\n", time);
    } else {
      fprintf(stderr, ") = %.1f FAILED %d\n", time, error);
    }

    if ((run == 0) || (mintime > runtime)) mintime = runtime;
  }

  fprintf(stderr, "mintime: %d ms\n", mintime);

  // print output
//  for (i = 0; i < nbodies; i++) {
    printf("%.2e %.2e %.2e\n", posx[i], posy[i], posz[i]);
//  }

  free(mass);
  free(posx);
  free(posy);
  free(posz);
  free(velx);
  free(vely);
  free(velz);

  hipFree(errl);
  hipFree(childl);
  hipFree(massl);
  hipFree(posxl);
  hipFree(posyl);
  hipFree(poszl);
  hipFree(countl);
  hipFree(startl);

  hipFree(maxxl);
  hipFree(maxyl);
  hipFree(maxzl);
  hipFree(minxl);
  hipFree(minyl);
  hipFree(minzl);

  return 0;
}
